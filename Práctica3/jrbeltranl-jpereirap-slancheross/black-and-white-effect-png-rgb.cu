// Tomado y adaptado de http://zarb.org/~gc/html/libpng.html

#include <unistd.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <stdarg.h>
#include <sys/time.h>
#include <omp.h>
#include <png.h>
#include <sched.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>


void abort_(const char * s, ...)
{
        va_list args;
        va_start(args, s);
        vfprintf(stderr, s, args);
        fprintf(stderr, "\n");
        va_end(args);
        abort();
}

int x, y;

int width, height;
int * d_width, *d_height;
png_byte color_type;
png_byte bit_depth;

png_structp png_ptr;
png_infop info_ptr;
int number_of_passes;
png_bytep * row_pointers;
struct timeval tval_before, tval_after, tval_result, tval_before1, tval_after1, tval_result1;
int * image_rgb_averages;
int n;
int * d_n;

void read_png_file(char* file_name)
{
        gettimeofday(&tval_before, NULL);
        
        
        char header[8];    // 8 is the maximum size that can be checked

        // Lee el archivo y verifica si es un PNG
        FILE *fp = fopen(file_name, "rb");
        if (!fp)
                abort_("[read_png_file] File %s could not be opened for reading", file_name);
        fread(header, 1, 8, fp);
        // if (png_sig_cmp(header, 0, 8))
        //         abort_("[read_png_file] File %s is not recognized as a PNG file", file_name);


        //Inicializa variables necesarias para libpng
        png_ptr =   png_create_read_struct (PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);

        if (!png_ptr)
                abort_("[read_png_file] png_create_read_struct failed");

        info_ptr = png_create_info_struct(png_ptr);
        if (!info_ptr)
                abort_("[read_png_file] png_create_info_struct failed");

        if (setjmp(png_jmpbuf(png_ptr)))
                abort_("[read_png_file] Error during init_io");

        //Inicializa el input/output para el archivo PNG
        png_init_io(png_ptr, fp);
        png_set_sig_bytes(png_ptr, 8);

        //Lee la información anterior a los datos de los píxeles como tal
        png_read_info(png_ptr, info_ptr);

        //Almacena información del archivo PNG 
        width = png_get_image_width(png_ptr, info_ptr);
        height = png_get_image_height(png_ptr, info_ptr);
        color_type = png_get_color_type(png_ptr, info_ptr);
        bit_depth = png_get_bit_depth(png_ptr, info_ptr);

        number_of_passes = png_set_interlace_handling(png_ptr);
        png_read_update_info(png_ptr, info_ptr);


        // Lectura del archivo PNG
        if (setjmp(png_jmpbuf(png_ptr)))
                abort_("[read_png_file] Error during read_image");

        // Reserva el espacio necesario para almacenar los datos del archivo PNG por filas
        row_pointers = (png_bytep*) malloc(sizeof(png_bytep) * height);
        for (y=0; y<height; y++)
                row_pointers[y] = (png_byte*) malloc(png_get_rowbytes(png_ptr,info_ptr));

        png_read_image(png_ptr, row_pointers);
        
        fclose(fp);
        gettimeofday(&tval_after, NULL);
        timersub(&tval_after, &tval_before, &tval_result);
        //printf("BnW Read: %ld.%06ld\n", (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);
}

int * image_to_array(int width, int height){

        static int image_as_array[24883200];
        int x = 0;
        int y = 0;
        int k = 0;
        int counter = 0;
        png_byte *row;
        png_byte *ptr;


        for (y=0; y<height-1; y++) {

                for (x=0; x<width; x++) {
                
                        row             = row_pointers[y];
                        ptr             = &(row[x*3]);
                        
                        for(k=0; k<3; k++){
                                image_as_array[counter] = ptr[k];
                                counter++;
                        }
                }
        }      
        return image_as_array;
}

void write_png_file(char* file_name)
{
        gettimeofday(&tval_before, NULL);
        // Crea el archivo
        FILE *fp = fopen(file_name, "wb");
        if (!fp)
                abort_("[write_png_file] File %s could not be opened for writing", file_name);


        //Inicializa variables necesarias para libpng
        png_ptr = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);

        if (!png_ptr)
                abort_("[write_png_file] png_create_write_struct failed");

        info_ptr = png_create_info_struct(png_ptr);
        if (!info_ptr)
                abort_("[write_png_file] png_create_info_struct failed");

        if (setjmp(png_jmpbuf(png_ptr)))
                abort_("[write_png_file] Error during init_io");

        png_init_io(png_ptr, fp);


        // Escribe el header del archivo PNG
        if (setjmp(png_jmpbuf(png_ptr)))
                abort_("[write_png_file] Error during writing header");

        png_set_IHDR(png_ptr, info_ptr, width, height,
                     bit_depth, color_type, PNG_INTERLACE_NONE,
                     PNG_COMPRESSION_TYPE_BASE, PNG_FILTER_TYPE_BASE);

        png_write_info(png_ptr, info_ptr);


        // Escribe los bytes del archivo PNG
        if (setjmp(png_jmpbuf(png_ptr)))
                abort_("[write_png_file] Error during writing bytes");

        png_write_image(png_ptr, row_pointers);


        // Termina la escritura
        if (setjmp(png_jmpbuf(png_ptr)))
                abort_("[write_png_file] Error during end of write");

        png_write_end(png_ptr, NULL);

        // Libera el espacio reservado previamente
        for (y=0; y<height; y++)
                free(row_pointers[y]);
        free(row_pointers);

        fclose(fp);

        gettimeofday(&tval_after, NULL);
        timersub(&tval_after, &tval_before, &tval_result);
        //printf("BnW Write: %ld.%06ld\n", (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);
}


__global__ void process_file(int * d_image_as_array, int * d_image_rgb_averages, int * d_width, int * d_height, int * d_n)
{
        // Se realizan los cambios deseados en la imagen
        //Verificar los datos recibidos
        //printf("Width = %d , Height = %d ", *d_width, *d_height);
        //printf("n = %d \n", *d_n );
        int array_size = (*d_width)*(*d_height-1)*3;
        int block_size = array_size / *d_n;
        int counter = (threadIdx.x * block_size) / 3;
        //printf("Thread number %d working from %d to %d, storing initiates at %d \n", threadIdx.x, threadIdx.x * block_size, (threadIdx.x*block_size) + block_size, counter );
        

        for (int i=(threadIdx.x * block_size); i<((threadIdx.x*block_size) + block_size); i=i+3) {
                d_image_rgb_averages[counter] = (d_image_as_array[i] + d_image_as_array[i+1] + d_image_as_array[i+2]) / 3;
                counter++;
        }
        
}

void from_array_to_bnw_png(int image_rgb_averages[])
{ 
        // Se toman los valores de promedio RGB y se guardan en la estructura pngbyte nuevamente
        int x = 0;
        int y = 0;
        int channels = 3;
        png_byte *row, *ptr;
        int counter = 0;

        for (y=0; y<height-1; y++) {

                for (x=0; x<width; x++) {

                        row             = row_pointers[y];
                        ptr             = &(row[x*channels]);
                        
                        ptr[0]  = image_rgb_averages[counter];
                        ptr[1]  = image_rgb_averages[counter];
                        ptr[2]  = image_rgb_averages[counter];
                        counter++;
                }
        }
}

int main(int argc, char **argv)
{       
        // Verifica los parámetros para ejecutar el programa
        if (argc != 4)
                abort_("Uso: ./Nombre_del_Programa <file_in> <file_out> <num_hilos>");

        // Timer de inicio total
        gettimeofday(&tval_before1, NULL);
        n = atoi(argv[3]);

        // Lee la imagen y pasa los datos a un array para pasar este parámetro al kernel
        read_png_file(argv[1]);

        printf("Para la imagen de resolución %d x %d, con %d hilos: \n", width, height, n);
        int * image_as_array;
        int * d_image_as_array;
        image_as_array = image_to_array(width, height);
        

        // Creamos un array que va a recibir la respuesta del kernel y su copia para el device
        int * d_image_rgb_averages;
        int image_rgb_averages_size = sizeof(int) * width*height;
        image_rgb_averages = (int *)malloc(image_rgb_averages_size);

        // CUDA
        // Tiempo de inicio para el procesamiento de la imagen
        gettimeofday(&tval_before, NULL);
        // Reservar el espacio para las copias en el device
        int int_size = sizeof(int);
        int image_as_array_size = sizeof(int)*width*height*3;
        
        hipMalloc((void **)&d_image_as_array, image_as_array_size);
        hipMalloc((void **)&d_width, int_size);        
        hipMalloc((void **)&d_height, int_size);
        hipMalloc((void **)&d_n, int_size);
        hipMalloc((void **)&d_image_rgb_averages, image_rgb_averages_size);

        // Copiar los inputs al device
        hipMemcpy(d_image_as_array, image_as_array, image_as_array_size, hipMemcpyHostToDevice);
        hipMemcpy(d_width, &width, int_size, hipMemcpyHostToDevice);
        hipMemcpy(d_height, &height, int_size, hipMemcpyHostToDevice); 
        hipMemcpy(d_n, &n, int_size, hipMemcpyHostToDevice);        

        // Lanzar el kernel
        process_file<<<1,n>>>(d_image_as_array, d_image_rgb_averages, d_width, d_height, d_n);

        // Copiar los resultados de vuelta al host
        hipMemcpy(image_rgb_averages, d_image_rgb_averages, image_rgb_averages_size, hipMemcpyDeviceToHost);              
        // Para verificar los valores devueltos
        // for (int i =0; i<8294400; i++){
        //         printf("%d, ", image_rgb_averages[i]);
        // }

        // Tiempo de fin para el procesamiento de la imagen
        gettimeofday(&tval_after, NULL);
        timersub(&tval_after, &tval_before, &tval_result);
        printf("BnW Process: %ld.%06ld\n", (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);

        // Se pasa del array al pngbyte nuevamente
        from_array_to_bnw_png(image_rgb_averages);

        // Escritura de la imagen con los resultados
        write_png_file(argv[2]);

        // Tiempo de fin total
        gettimeofday(&tval_after1, NULL);
        timersub(&tval_after1, &tval_before1, &tval_result1);
        printf("Tiempo de ejecución total: %ld.%06ld\n \n", (long int)tval_result1.tv_sec, (long int)tval_result1.tv_usec);

        // Limpieza
        hipFree(d_width);
        hipFree(d_height);
        hipFree(d_n);
        hipFree(d_image_as_array);
        hipFree(d_image_rgb_averages);
        
        return 0;
}
